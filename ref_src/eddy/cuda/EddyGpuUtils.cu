/////////////////////////////////////////////////////////////////////
///
/// \file EddyGpuUtils.cu
/// \brief Definitions of static class with collection of GPU routines used in the eddy project
///
/// \author Jesper Andersson & Moises Hernandez
/// \version 1.0b, Nov., 2012.
/// \Copyright (C) 2012 University of Oxford 
///


#include <cstdlib>
#include <string>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#pragma push
#pragma diag_suppress = code_is_unreachable 
#pragma diag_suppress = expr_has_no_effect  
#include "newmat.h"
#include "newimage/newimageall.h"
#pragma pop
#include "miscmaths/miscmaths.h"
#include "EddyInternalGpuUtils.h"
#include "EddyHelperClasses.h"
#include "DiffusionGP.h"
#include "b0Predictor.h"
#include "ECScanClasses.h"
#include "EddyUtils.h"
#include "EddyGpuUtils.h"
#include "EddyKernels.h"

using namespace EDDY;

void EddyGpuUtils::InitGpu(bool verbose) EddyTry
{
  static bool initialized=false;
  if (!initialized) {
    initialized=true;
    int device;
    hipError_t ce;
    if ((ce = hipGetDevice(&device)) != hipSuccess) {
      if (ce == hipErrorInvalidValue) throw EddyException("EddyGpuUtils::InitGpu: cudeGetDevice returned an error code hipErrorInvalidValue");
      else throw EddyException("EddyGpuUtils::InitGpu: cudeGetDevice returned an unknown error code");
    }    
    if (verbose) printf("\n...................Allocated GPU # %d...................\n", device); 
    int *q;
    if ((ce = hipMalloc((void **)&q, sizeof(int))) != hipSuccess) {
      throw EddyException("EddyGpuUtils::InitGpu: cudeMalloc returned an error when trying to allocate device memory");
    }
    hipFree(q);
    EddyKernels::CudaSync("EddyGpuUtils::InitGpu");
  }
} EddyCatch


std::shared_ptr<DWIPredictionMaker> EddyGpuUtils::LoadPredictionMaker(
								      const EddyCommandLineOptions& clo,
								      ScanType                      st,
								      const ECScanManager&          sm,
								      unsigned int                  iter,
								      float                         fwhm,
								      
								      NEWIMAGE::volume<float>&      mask,
								      
								      bool                          use_orig) EddyTry
{
  std::shared_ptr<DWIPredictionMaker>  pmp;                                 
  if (st==DWI) { 
    std::shared_ptr<KMatrix> K;
    if (clo.CovarianceFunction() == Spherical) K = std::shared_ptr<SphericalKMatrix>(new SphericalKMatrix(clo.DontCheckShelling()));
    else if (clo.CovarianceFunction() == Exponential) K = std::shared_ptr<ExponentialKMatrix>(new ExponentialKMatrix(clo.DontCheckShelling()));
    else if (clo.CovarianceFunction() == NewSpherical) K = std::shared_ptr<NewSphericalKMatrix>(new NewSphericalKMatrix(clo.DontCheckShelling()));
    else throw EddyException("LoadPredictionMaker: Unknown covariance function");
    std::shared_ptr<HyParCF> hpcf;
    std::shared_ptr<HyParEstimator> hpe;
    if (clo.HyperParFixed()) hpe = std::shared_ptr<FixedValueHyParEstimator>(new FixedValueHyParEstimator(clo.HyperParValues()));
    else {
      if (clo.HyParCostFunction() == CC) hpe = std::shared_ptr<CheapAndCheerfulHyParEstimator>(new CheapAndCheerfulHyParEstimator(clo.NVoxHp(),clo.InitRand()));
      else {
	if (clo.HyParCostFunction() == MML) hpcf = std::shared_ptr<MMLHyParCF>(new MMLHyParCF);
	else if (clo.HyParCostFunction() == CV) hpcf = std::shared_ptr<CVHyParCF>(new CVHyParCF);
	else if (clo.HyParCostFunction() == GPP) hpcf = std::shared_ptr<GPPHyParCF>(new GPPHyParCF);
	else throw EddyException("LoadPredictionMaker: Unknown hyperparameter cost-function");
	hpe = std::shared_ptr<FullMontyHyParEstimator>(new FullMontyHyParEstimator(hpcf,clo.HyParFudgeFactor(),clo.NVoxHp(),clo.InitRand(),clo.VeryVerbose()));
      }
    }
    pmp = std::shared_ptr<DWIPredictionMaker>(new DiffusionGP(K,hpe));  
  }
  else pmp = std::shared_ptr<DWIPredictionMaker>(new b0Predictor);          
  pmp->SetNoOfScans(sm.NScans(st));
  mask = sm.Scan(0,ANY).GetIma(); EddyUtils::SetTrilinearInterp(mask); mask = 1.0;

  EddyGpuUtils::InitGpu();
  EddyInternalGpuUtils::load_prediction_maker(clo,st,sm,iter,fwhm,use_orig,sm.GetPolation(),pmp,mask);

  if (clo.DebugLevel() > 2 && st==DWI) {
    char fname[256];
    sprintf(fname,"EDDY_DEBUG_K_Mat_Data_%02d",iter);
    pmp->WriteMetaData(fname);
  }

  return(pmp);
} EddyCatch



NEWIMAGE::volume<float> EddyGpuUtils::GetUnwarpedScan(
						      const EDDY::ECScan&                               scan,
						      std::shared_ptr<const NEWIMAGE::volume<float> >   susc,
						      bool                                              use_orig,
						      const EDDY::PolationPara&                         pp,
						      
						      NEWIMAGE::volume<float>                           *omask) EddyTry
{
  EDDY::CudaVolume cuda_susc;
  if (susc) cuda_susc = *susc;
  EDDY::CudaVolume empty;
  EDDY::CudaVolume uwscan(scan.GetIma(),false);
  if (omask) {
    EDDY::CudaVolume tmpmask(*omask,false);
    EddyInternalGpuUtils::get_unwarped_scan(scan,cuda_susc,empty,true,use_orig,pp,uwscan,tmpmask);
    *omask = tmpmask.GetVolume();
  }
  else {
    EDDY::CudaVolume tmpmask;
    EddyInternalGpuUtils::get_unwarped_scan(scan,cuda_susc,empty,true,use_orig,pp,uwscan,tmpmask);
  }
  return(uwscan.GetVolume());
} EddyCatch

NEWIMAGE::volume<float> EddyGpuUtils::GetUnwarpedScan(
						      const EDDY::ECScan&                               scan,
						      std::shared_ptr<const NEWIMAGE::volume<float> >   susc,
						      const NEWIMAGE::volume<float>&                    pred,
						      bool                                              use_orig,
						      const EDDY::PolationPara&                         pp,
						      
						      NEWIMAGE::volume<float>                           *omask) EddyTry
{
  if (!scan.IsSliceToVol()) {
    cout << "EddyGpuUtils::GetUnwarpedScan: Warning, it does not make sense to supply pred for volumetric resampling" << endl;
  }
  if (pp.GetS2VInterp() != NEWIMAGE::spline) {
    throw EddyException("EddyGpuUtils::GetUnwarpedScan: use of prediction cannot be combined with trilinear interpolation");
  }
  EDDY::CudaVolume cuda_susc;
  if (susc) cuda_susc = *susc;
  EDDY::CudaVolume uwscan(scan.GetIma(),false);
  EDDY::CudaVolume cuda_pred = pred;
  if (omask) {
    EDDY::CudaVolume tmpmask(*omask,false);
    EddyInternalGpuUtils::get_unwarped_scan(scan,cuda_susc,cuda_pred,true,use_orig,pp,uwscan,tmpmask);
    *omask = tmpmask.GetVolume();
  }
  else {
    EDDY::CudaVolume tmpmask;
    EddyInternalGpuUtils::get_unwarped_scan(scan,cuda_susc,cuda_pred,true,use_orig,pp,uwscan,tmpmask);
  }
  return(uwscan.GetVolume());
} EddyCatch

NEWIMAGE::volume<float> EddyGpuUtils::GetVolumetricUnwarpedScan(
								const EDDY::ECScan&                               scan,
								std::shared_ptr<const NEWIMAGE::volume<float> >   susc,
								bool                                              use_orig,
								const EDDY::PolationPara&                         pp,
								
								NEWIMAGE::volume<float>                           *omask,
								NEWIMAGE::volume4D<float>                         *deriv) EddyTry
{
  EDDY::CudaVolume cuda_susc;
  if (susc) cuda_susc = *susc;
  EDDY::CudaVolume empty;
  EDDY::CudaVolume uwscan(scan.GetIma(),false);
  if (omask && deriv) {
    EDDY::CudaVolume tmpmask(*omask,false);
    EDDY::CudaVolume4D tmpderiv(*deriv,false);
    EddyInternalGpuUtils::get_volumetric_unwarped_scan(scan,cuda_susc,true,use_orig,pp,uwscan,tmpmask,tmpderiv);
    *omask = tmpmask.GetVolume();
    *deriv = tmpderiv.GetVolume();
  }
  else if (omask) {
    EDDY::CudaVolume tmpmask(*omask,false);
    EDDY::CudaVolume4D tmpderiv;
    EddyInternalGpuUtils::get_volumetric_unwarped_scan(scan,cuda_susc,true,use_orig,pp,uwscan,tmpmask,tmpderiv);
    *omask = tmpmask.GetVolume();
  }
  else if (deriv) {
    EDDY::CudaVolume tmpmask;
    EDDY::CudaVolume4D tmpderiv(*deriv,false);
    EddyInternalGpuUtils::get_volumetric_unwarped_scan(scan,cuda_susc,true,use_orig,pp,uwscan,tmpmask,tmpderiv);
    *deriv = tmpderiv.GetVolume();
  }
  else {
    EDDY::CudaVolume tmpmask;
    EDDY::CudaVolume4D tmpderiv;
    EddyInternalGpuUtils::get_volumetric_unwarped_scan(scan,cuda_susc,true,use_orig,pp,uwscan,tmpmask,tmpderiv);
  }
  return(uwscan.GetVolume());
} EddyCatch

void EddyGpuUtils::GetMotionCorrectedScan(
					  const EDDY::ECScan&       scan,
					  bool                      use_orig,
					  
					  NEWIMAGE::volume<float>&  ovol,
					  
					  NEWIMAGE::volume<float>   *omask) EddyTry
{
  EDDY::CudaVolume covol(scan.GetIma(),false);
  EDDY::CudaVolume comask;
  if (omask) {comask.SetHdr(covol); comask = 1.0; }
  EddyInternalGpuUtils::get_motion_corrected_scan(scan,use_orig,covol,comask);
  ovol = covol.GetVolume();
  if (omask) *omask = comask.GetVolume();
} EddyCatch

NEWIMAGE::volume<float> EddyGpuUtils::TransformModelToScanSpace(const EDDY::ECScan&                               scan,
								const NEWIMAGE::volume<float>&                    mima,
								std::shared_ptr<const NEWIMAGE::volume<float> >   susc,
								bool                                              jacmod) EddyTry
{
  EDDY::CudaVolume mima_gpu = mima;
  EDDY::CudaVolume susc_gpu;
  if (susc != nullptr) susc_gpu = *susc;
  EDDY::CudaVolume mios(mima,false);
  EDDY::CudaVolume mask(mima,false); mask=1.0;
  EDDY::CudaVolume jac(mima,false);
  EDDY::CudaVolume4D skrutt4D;
  
  EddyInternalGpuUtils::transform_model_to_scan_space(mima_gpu,scan,susc_gpu,jacmod,mios,mask,jac,skrutt4D);
  
  return(mios.GetVolume());
} EddyCatch

NEWIMAGE::volume4D<float> EddyGpuUtils::DerivativesForModelToScanSpaceTransform(const EDDY::ECScan&                               scan,
										const NEWIMAGE::volume<float>&                    mima,
										std::shared_ptr<const NEWIMAGE::volume<float> >   susc) EddyTry
{
  EDDY::CudaVolume mima_gpu = mima;
  EDDY::CudaVolume susc_gpu;
  if (susc != nullptr) susc_gpu = *susc;
  EDDY::CudaVolume4D derivs(mima,scan.NDerivs(),false);
  EddyInternalGpuUtils::get_partial_derivatives_in_scan_space(mima_gpu,scan,susc_gpu,EDDY::ALL,derivs);
  return(derivs.GetVolume());
} EddyCatch

NEWIMAGE::volume4D<float> EddyGpuUtils::DirectDerivativesForModelToScanSpaceTransform(const EDDY::ECScan&                               scan,
										      const NEWIMAGE::volume<float>&                    mima,
										      std::shared_ptr<const NEWIMAGE::volume<float> >   susc) EddyTry
{
  EDDY::CudaVolume mima_gpu = mima;
  EDDY::CudaVolume susc_gpu;
  if (susc != nullptr) susc_gpu = *susc;
  EDDY::CudaVolume4D derivs(mima,scan.NDerivs(),false);
  EddyInternalGpuUtils::get_direct_partial_derivatives_in_scan_space(mima_gpu,scan,susc_gpu,EDDY::ALL,derivs);
  return(derivs.GetVolume());
} EddyCatch

NEWIMAGE::volume<float> EddyGpuUtils::Smooth(const NEWIMAGE::volume<float>&  ima,
					     float                           fwhm) EddyTry
{
  EDDY::CudaVolume cuda_ima(ima,true);
  cuda_ima.Smooth(fwhm);
  return(cuda_ima.GetVolume());
} EddyCatch

DiffStatsVector EddyGpuUtils::DetectOutliers(
					     const EddyCommandLineOptions&             clo,
					     ScanType                                  st,
					     const std::shared_ptr<DWIPredictionMaker> pmp,
					     const NEWIMAGE::volume<float>&            mask,
					     const ECScanManager&                      sm,
					     
					     ReplacementManager&                       rm) EddyTry
{
  EddyGpuUtils::InitGpu();
  DiffStatsVector  dsv(sm.NScans(st));
  EddyInternalGpuUtils::detect_outliers(clo,st,pmp,mask,sm,rm,dsv);
  return(dsv);
} EddyCatch

void EddyGpuUtils::ReplaceOutliers(
				   const EddyCommandLineOptions&             clo,
				   ScanType                                  st,
				   const std::shared_ptr<DWIPredictionMaker> pmp,
				   const NEWIMAGE::volume<float>&            mask,
				   const ReplacementManager&                 rm,
				   bool                                      add_noise,
				   
				   ECScanManager&                            sm) EddyTry
{
  EddyGpuUtils::InitGpu();
  EddyInternalGpuUtils::replace_outliers(clo,st,pmp,mask,rm,add_noise,sm);
} EddyCatch

double EddyGpuUtils::MovAndECParamUpdate(
					 const NEWIMAGE::volume<float>&                    pred,
					 std::shared_ptr<const NEWIMAGE::volume<float> >   susc,
					 const NEWIMAGE::volume<float>&                    pmask,
					 bool                                              cbs,
					 float                                             fwhm,
					 const EDDY::PolationPara&                         pp,
					 
					 EDDY::ECScan&                                     scan) EddyTry
{
  EddyGpuUtils::InitGpu();
  return(EddyInternalGpuUtils::param_update(pred,susc,pmask,EDDY::ALL,cbs,fwhm,pp,0,0,0,scan,NULL));
} EddyCatch

double EddyGpuUtils::MovAndECParamUpdate(
					 const NEWIMAGE::volume<float>&                    pred,
					 std::shared_ptr<const NEWIMAGE::volume<float> >   susc,
					 const NEWIMAGE::volume<float>&                    pmask,
					 bool                                              cbs,
					 float                                             fwhm,
					 const EDDY::PolationPara&                         pp,
					 
					 unsigned int                                      scindex,
					 unsigned int                                      iter,
					 unsigned int                                      level,
					 
					 EDDY::ECScan&                                     scan) EddyTry
{
  EddyGpuUtils::InitGpu();
  return(EddyInternalGpuUtils::param_update(pred,susc,pmask,EDDY::ALL,cbs,fwhm,pp,scindex,iter,level,scan,NULL));
} EddyCatch

